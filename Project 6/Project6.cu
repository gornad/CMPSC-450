
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define N 523216
#define B 1024
#define NUM_BANKS 16
#define LOG_NUM_BANKS 4
#define CONFLICT_FREE_OFFSET(n) \
    ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))
	

__global__ void scan(double *g_odata, double *g_idata, int n);
__global__ void prescan(double *g_odata, double *g_idata, double *blocksum, int n, int sharedmemory, int maxblocksize);
__global__ void gpusummation(double *g_odata, double *g_idata, double *blocksum, int n);

void bestscan(double *g_odata, double *g_idata, int n);
void scanCPU(double *f_out, double *f_in, int i_n);

double myDiffTime(struct timeval &start, struct timeval &end)
{
	double d_start, d_end;
	d_start = (double)(start.tv_sec + start.tv_usec/1000000.0);
	d_end = (double)(end.tv_sec + end.tv_usec/1000000.0);
	return (d_end - d_start);
}

int main() 
{
	double a[N], c[N], g[N];
	timeval start, end;

	double *dev_a, *dev_g;
	int size = N * sizeof(double);
	
	double d_gpuTime, d_cpuTime;

	// initialize matrices a 
	for (int i = 0; i < N; i++)
	{
		a[i] = (double)(rand() % 1000000) / 1000.0;
		//a[i] = 1;
		//printf("a[%i] = %f\n", i, a[i]);
	}
	// initialize a and b matrices here
	hipMalloc((void **) &dev_a, size);
	hipMalloc((void **) &dev_g, size);

	
	gettimeofday(&start, NULL);

	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);

	bestscan(dev_g, dev_a, N);
	hipDeviceSynchronize();

	hipMemcpy(g, dev_g, size, hipMemcpyDeviceToHost);

	gettimeofday(&end, NULL);
	d_gpuTime = myDiffTime(start, end);

	gettimeofday(&start, NULL);
	scanCPU(c, a, N);
	
	gettimeofday(&end, NULL);
	d_cpuTime = myDiffTime(start, end);
	

	hipFree(dev_a); hipFree(dev_g);

	for (int i = 0; i < N; i++)
	{
		printf("c[%i] = %0.3f, g[%i] = %0.3f\n", i, c[i], i, g[i]);
	}

	printf("GPU Time for scan size %i: %f\n", N, d_gpuTime);
	printf("CPU Time for scan size %i: %f\n", N, d_cpuTime);
	
	// Built bank functionality from scratch
	// Changed float to double for better accuracy
}

void bestscan(double *g_odata, double *g_idata, int n) {

	int size = n * sizeof(double);
	double* blocksum;
	int maxblocksize = B;
	int sharedmemory = maxblocksize + ((maxblocksize-1)>> LOG_NUM_BANKS);
	int num_thread_blocks = (n/B);
	// No shortage of blocks
	if((N % B) != 0)
		num_thread_blocks++;
	
	hipMalloc(&blocksum, sizeof(double)*num_thread_blocks); //array of sums
	hipMemset(blocksum, 0, sizeof(double)*num_thread_blocks);
	
	hipMemset(g_odata, 0, size);
	
	prescan<<<num_thread_blocks,B/2,sharedmemory*sizeof(double)>>>(g_odata, g_idata, blocksum, n, sharedmemory, maxblocksize);
	
	if (num_thread_blocks <= maxblocksize)
	{
		//Only one block needed
		double* fillblocksum;
		hipMalloc(&fillblocksum, sizeof(double));
		hipMemset(fillblocksum, 0, sizeof(double));
		prescan<<<1, B/2, sharedmemory*sizeof(double)>>>(blocksum, blocksum, fillblocksum, num_thread_blocks, sharedmemory, maxblocksize);
		hipFree(fillblocksum);
	}

	else
	{
		// Recursive function call for summation of multiple blocks
		double* blocksum2;
		hipMalloc(&blocksum2, sizeof(double)*num_thread_blocks);
		hipMemcpy(blocksum2, blocksum, sizeof(double)*num_thread_blocks, hipMemcpyDeviceToDevice);
		bestscan(blocksum, blocksum2, num_thread_blocks);
		hipFree(blocksum2);
	}
	
	gpusummation<<<num_thread_blocks, B/2>>>(g_odata, g_odata, blocksum, n);	
	
	hipFree(blocksum);
}

__global__ void gpusummation(double *g_odata, double *g_idata, double* blocksum, int n) {
	
	// SUMMATION
	
	double sum = blocksum[blockIdx.x];
	int idx = 2*blockIdx.x*blockDim.x + threadIdx.x;
	
	if (n > idx)
	{
		g_odata[idx] = g_odata[idx] + sum;
		if (n > idx + blockDim.x)
			g_odata[idx + blockDim.x] = g_odata[idx+blockDim.x]+sum;
	}
}

//Heavily inspired by CUDA Gems3 book with slight changes to support bigger array sizes

__global__ void prescan(double *g_odata, double *g_idata, double *blocksum, int n, int sharedmemory, int maxblocksize) 
{ 
	extern  __shared__  double temp[];
	// allocated on invocation 
	int thid = threadIdx.x; 
	int offset = 1; 
	
	//A
	int ai = thid;
	int bi = thid + blockDim.x;
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
	int idx = maxblocksize * blockIdx.x + threadIdx.x;
	
	temp[thid] = 0;
	temp[thid+blockDim.x] = 0;

	if (thid + maxblocksize < sharedmemory)
		temp[thid + maxblocksize] = 0;

	__syncthreads();
	
	//Fill temporary shared buffer
	if (n > idx){
	
		temp[ai + bankOffsetA] = g_idata[idx];
		if (n > idx + blockDim.x)
			temp[bi + bankOffsetB] = g_idata[idx + blockDim.x];
	}
	
	for (int d = maxblocksize>>1; d > 0; d >>= 1) 
	// build sum in place up the tree 
    	{ 
        	__syncthreads(); 
		if (thid < d)    
        	{ 
			
			//B
				int ai = offset*(2*thid+1)-1;
				int bi = offset*(2*thid+2)-1;
				ai += CONFLICT_FREE_OFFSET(ai);
				bi += CONFLICT_FREE_OFFSET(bi);
		    	temp[bi] += temp[ai];         
				
			} 
        	offset *= 2; 
    	} 

		if (thid==0) { 
			blocksum[blockIdx.x] = temp[maxblocksize - 1 + CONFLICT_FREE_OFFSET(maxblocksize - 1)];
			temp[maxblocksize - 1 + CONFLICT_FREE_OFFSET(maxblocksize - 1)] = 0;
		}

	// clear the last element 
	for (int d = 1; d < maxblocksize; d *= 2) 
	// traverse down tree & build scan 
    	{ 
        	offset >>= 1; 
        	__syncthreads(); 
		if (thid < d) 
        	{ 
			
				//D
				int ai = offset*(2*thid+1)-1;
				int bi = offset*(2*thid+2)-1;
				ai += CONFLICT_FREE_OFFSET(ai);
				bi += CONFLICT_FREE_OFFSET(bi);
				
				
				double t   = temp[ai]; 
				temp[ai]  = temp[bi]; 
				temp[bi] += t; 
        	} 
    	} 
    	__syncthreads(); 
		
	//Output shared buffer
	if (n > idx)
	{
		g_odata[idx] = temp[ai + bankOffsetA];
		if (n > idx + blockDim.x)
			g_odata[idx + blockDim.x] = temp[bi + bankOffsetB];
	}
}
 
void scanCPU(double *f_out, double *f_in, int i_n)
{
	f_out[0] = 0;
	for (int i = 1; i < i_n; i++)
		f_out[i] = f_out[i-1] + f_in[i-1];

}
